#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

#include "optixRaycastingKernels.h"

#include <sutil/vec_math.h>


inline int idivCeil(int x, int y)
{
    return (x + y - 1) / y;
}


__global__ void createRaysOrthoKernel(Ray* rays, int width, int height, float x0, float y0, float z, float dx, float dy)
{
    const int rayx = threadIdx.x + blockIdx.x * blockDim.x;
    const int rayy = threadIdx.y + blockIdx.y * blockDim.y;
    if (rayx >= width || rayy >= height)
        return;

    const int idx = rayx + rayy * width;
    rays[idx].origin = make_float3(x0 + rayx * dx, y0 + rayy * dy, z);
    rays[idx].tmin = 0.0f;
    rays[idx].dir = make_float3(0, 0, 1);
    rays[idx].tmax = 1e34f;
}


// Note: uses left handed coordinate system
void createRaysOrthoOnDevice(Ray* rays_device, int width, int height, float3 bbmin, float3 bbmax, float padding)
{
    const float3 bbspan = bbmax - bbmin;
    float        dx = bbspan.x * (1 + 2 * padding) / width;
    float        dy = bbspan.y * (1 + 2 * padding) / height;
    float        x0 = bbmin.x - bbspan.x * padding + dx / 2;
    float        y0 = bbmin.y - bbspan.y * padding + dy / 2;
    float        z = bbmin.z - fmaxf(bbspan.z, 1.0f) * .001f;

    dim3 blockSize(32, 16);
    dim3 gridSize(idivCeil(width, blockSize.x), idivCeil(height, blockSize.y));
    createRaysOrthoKernel << <gridSize, blockSize >> > (rays_device, width, height, x0, y0, z, dx, dy);
}


__global__ void translateRaysKernel(Ray* rays, int count, float3 offset)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= count)
        return;

    rays[idx].origin = rays[idx].origin + offset;
}


void translateRaysOnDevice(Ray* rays_device, int count, float3 offset)
{
    const int blockSize = 512;
    const int blockCount = idivCeil(count, blockSize);
    translateRaysKernel << <blockCount, blockSize >> > (rays_device, count, offset);
}


__global__ void shadeHitsKernel(float3* image, int count, const Hit* hits)
{

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= count)
        return;

    const float3 backgroundColor = make_float3(0.2f, 0.2f, 0.2f);
    if (hits[idx].t < 0.0f)
    {
        image[idx] = backgroundColor;
    }
    else
    {
        image[idx] = 0.5f * hits[idx].geom_normal + make_float3(0.5f, 0.5f, 0.5f);
    }
}


void shadeHitsOnDevice(float3* image_device, int count, const Hit* hits_device)
{
    const int blockSize = 512;
    const int blockCount = idivCeil(count, blockSize);
    shadeHitsKernel << <blockCount, blockSize >> > (image_device, count, hits_device);
}

