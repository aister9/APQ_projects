#include "hip/hip_runtime.h"
#include <optix.h>

#include "optixRaycasting.h"
#include "optixRaycastingKernels.h"

#include "cuda/LocalGeometry.h"
#include "cuda/whitted.h"

#include <sutil/vec_math.h>


extern "C" {
    __constant__ Params params;
}


extern "C" __global__ void __raygen__from_buffer()
{
    const uint3        idx = optixGetLaunchIndex();
    const uint3        dim = optixGetLaunchDimensions();
    const unsigned int linear_idx = idx.z * dim.y * dim.x + idx.y * dim.x + idx.x;

    unsigned int t, nx, ny, nz;
    Ray          ray = params.rays[linear_idx];
    optixTrace(params.handle, ray.origin, ray.dir, ray.tmin, ray.tmax, 0.0f, OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE, RAY_TYPE_RADIANCE, RAY_TYPE_COUNT, RAY_TYPE_RADIANCE, t, nx, ny, nz);

    Hit hit;
    hit.t = int_as_float(t);
    hit.geom_normal.x = int_as_float(nx);
    hit.geom_normal.y = int_as_float(ny);
    hit.geom_normal.z = int_as_float(nz);
    params.hits[linear_idx] = hit;
}


extern "C" __global__ void __miss__buffer_miss()
{
    optixSetPayload_0(float_as_int(-1.0f));
    optixSetPayload_1(float_as_int(1.0f));
    optixSetPayload_2(float_as_int(0.0f));
    optixSetPayload_3(float_as_int(0.0f));
}


extern "C" __global__ void __closesthit__buffer_hit()
{
    const unsigned int t = optixGetRayTmax();

    whitted::HitGroupData* rt_data = (whitted::HitGroupData*)optixGetSbtDataPointer();
    LocalGeometry          geom = getLocalGeometry(rt_data->geometry_data);

    // Set the hit data
    optixSetPayload_0(float_as_int(t));
    optixSetPayload_1(float_as_int(geom.N.x));
    optixSetPayload_2(float_as_int(geom.N.y));
    optixSetPayload_3(float_as_int(geom.N.z));
}


extern "C" __global__ void __anyhit__texture_mask()
{
    whitted::HitGroupData* rt_data = (whitted::HitGroupData*)optixGetSbtDataPointer();
    LocalGeometry          geom = getLocalGeometry(rt_data->geometry_data);

    float4 mask = tex2D<float4>(rt_data->material_data.pbr.base_color_tex, geom.UV.x, geom.UV.y);
    if (mask.x < 0.5f && mask.y < 0.5f)
    {
        optixIgnoreIntersection();
    }
}

